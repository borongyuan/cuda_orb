#include "hip/hip_runtime.h"
#include "Orb.h"
#include "FAST.cuh"
#include "Profiler.h"
#include <thread>

using namespace cv;
Orb::Orb() : rBRIEF()
{

}
void Orb::computeOrientation(cuArray<unsigned char>& frame, std::vector<float4>& corners, int width, int height)
{
	int cc = corners.size() < CORNER_LIMIT ? corners.size() : CORNER_LIMIT;
	AngleMap.upload(corners.data(), cc);
	ComputeOrientation << < corners.size() / 32, 32 >> > (frame, AngleMap, cc, width, height);
	AngleMap.download(corners.data(), cc);
}

std::vector<float4> Orb::detectKeypoints(cuArray<uchar>& ibuffer, cuArray<uchar>& aux, int thres, const int arc_length, const int width, const int height,const int limit, const int padding)
{
	std::vector<float4> corners;
	cv::Mat auxmat = cv::Mat(width, height, CV_8UC1);
	FAST << < dim3(width / FAST_TILE, height / FAST_TILE), dim3(FAST_TILE, FAST_TILE) >> > (ibuffer, aux, thres, arc_length, width, height);
	aux.download(auxmat.data);
	int x = padding + padding*width;
	for (uint i = padding; i < height - padding; ++i,x += padding * 2)
	{
		for (uint j = padding; j < width - padding; ++j,++x)
		{
			uint cvalue = auxmat.data[x];
			if (cvalue > 0)
			{
				corners.push_back({ (float)j,(float)i,0,0 });
			}
		}
	}
	if (limit>0)
	{
		AngleMap.upload(corners.data(), corners.size());
		FAST_Refine << < corners.size() / 32, 32 >> >(ibuffer, AngleMap, corners.size(), width, height);
		AngleMap.download(corners.data(), corners.size());
		std::sort(corners.begin(), corners.end(), [](float4& c1, float4& c2) {
			return c1.w > c2.w;
		});
		int minc = corners.size() >= limit ? limit : corners.size();
		std::vector<float4> strong = std::vector<float4>(corners.begin(), corners.begin() + minc);
		corners = strong;
	}
	computeOrientation(ibuffer, corners, width, height);
	return corners;
}


std::vector<float4> Orb::detectKeypoints(cv::Mat& grey, int thres, const int arc_length,const int limit, const int padding)
{
	int frameWidth = grey.cols, frameHeight = grey.rows;
	cuArray<uchar> gpuInputBuffer(frameWidth*frameHeight), gpuOutputBuffer(frameWidth*frameHeight);
	gpuInputBuffer.upload(grey.data);
	std::thread first([&] { boxFilter(grey, grey, -1, Size(5, 5)); });
	std::vector<float4> corners = detectKeypoints(gpuInputBuffer, gpuOutputBuffer, thres, arc_length, frameWidth, frameHeight,limit);
	first.join();
	return corners;
}

std::vector<BRIEF::BRIEF::Feature> Orb::extractFeatures(uint8_t** image, std::vector<float4> keypoints,MODE mode) const
{
	std::vector<cv::Point2d> corners;
	std::vector<float> angles;
	for (int i = 0; i < keypoints.size(); ++i)
	{
		corners.push_back(cv::Point2d(keypoints[i].x, keypoints[i].y));
		angles.push_back(keypoints[i].z);
	}
	if(mode == MODE::MODE_RBRIEF)
		return rBRIEF::extractFeatures(image, corners, angles);
	else
		return BRIEF::extractFeatures(image, corners);

}
