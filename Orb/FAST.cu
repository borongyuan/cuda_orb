#include "hip/hip_runtime.h"

#define TILE_RADIUS FAST_TILE/2
#include "FAST.cuh"
#include ""
__global__ void FAST(unsigned char* __restrict__ inputImage, unsigned char* __restrict__ cornerMap, const int threshold, const int width, const  int height)
{
	const int offsetX[27] = { 3, 3, 2, 1, 0, -1, -2, -3, -3, -3, -2, -1, 0, 1, 2, 3, 3, 3, 2, 1, 0, -1, -2, -3, -3, -3, -2 };
	const int offsetY[27] = { 0, 1, 2, 3, 3, 3, 2, 1, 0, -1, -2, -3, -3, -3, -2, -1, 0, 1, 2, 3, 3, 3, 2, 1, 0, -1, -2 };
	__shared__ int tile[FAST_TILE * 2 + 1][FAST_TILE * 2 + 1];
	int hblocks = width / FAST_TILE;
	int vblocks = height / FAST_TILE;
	int sourceX = blockIdx.x*blockDim.x + threadIdx.x;
	int sourceY = blockIdx.y*blockDim.y + threadIdx.y;
	int source = sourceX + sourceY*width;

	if (blockIdx.x > 0 && blockIdx.x < hblocks  && blockIdx.y>0 && blockIdx.y < vblocks)
	{
		for (int i = 0; i <= FAST_TILE; i += FAST_TILE)
			for (int j = 0; j <= FAST_TILE; j += FAST_TILE)
			{
				int xdestX = threadIdx.x + i;
				int xdestY = threadIdx.y + j;
				int xsourceX = blockIdx.x*blockDim.x + xdestX - TILE_RADIUS;
				int xsourceY = blockIdx.y*blockDim.y + xdestY - TILE_RADIUS;
				int xsource = xsourceX + xsourceY*width;
				tile[xdestX][xdestY] = inputImage[xsource];
			}
		__syncthreads();
		//FAST Algorithm
		int highCount = 0, lowCount = 0;
		int cX = threadIdx.x + TILE_RADIUS, cY = threadIdx.y + TILE_RADIUS;
		int center = tile[cX][cY];
		int t_low = (center < threshold) ? 0 : center - threshold;
		int t_high = (center > 255 - threshold) ? 255 : center + threshold;
		bool isCorner = false;
		bool CornerType = false;
		for (int i = 0; i < 27; ++i)
		{
			int x = offsetX[i] + cX, y = offsetY[i] + cY;
			highCount = (tile[x][y] > t_high) ? highCount + 1 : 0;
			lowCount = (tile[x][y] < t_low) ? lowCount + 1 : 0;
			if (highCount >= 12)
			{

				isCorner = true;
				CornerType = true;
			}
			else if (lowCount >=12)
			{
				isCorner = true;
				CornerType = false;
			}
		}
		cornerMap[source] = isCorner ? (CornerType ? 255 : 128) : 0;
	}
}




__device__ void localElMul(float in1[5][5], float in2[5][5], float out[5][5])
{
	for (int i = 0; i<5; ++i)
		for (int j = 0; j < 5; ++j)
		{
			out[i][j] = in1[i][j] * in2[i][j];
		}
}

__device__ void localConvolution(float roi[5][5], const float* k, int kdim, int roi_dim)
{

}


__global__ void FAST_Refine(unsigned char* __restrict__ inputImage, uint4* __restrict__ cornerMap,const int count, const int width, const  int height)
{
	int threadIndex = threadIdx.x + blockDim.x*blockIdx.x;
	uint4 cvector = cornerMap[threadIndex];
	

	const float gk3[] = { 0.25,0.5,0.25 };
	const int tile_size = 3;
	float roi[5][5];
	
	int ptr = cvector.x + cvector.y*width;
	for (int i = 0; i < 5; ++i)
	{
		for (int j = 0; j < 5; ++j)
		{
			roi[i][j] = inputImage[ptr];
			ptr += 1;
		}
		ptr += width - 5;
	}

	localConvolution(roi, gk3, 3, 5);
}



