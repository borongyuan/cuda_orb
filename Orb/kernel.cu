#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>
#include "Profiler.h"
#include "Memory.cuh"
#include "FAST.cuh"
#include "Orb.h"
#include <thread>
using namespace cv;
using namespace std;


Mat renderTrajectory(Mat& iframe)
{
	const int hframe_count = 8;
	static vector<Mat> history;
	history.push_back(iframe);
	Mat rframe(iframe.rows, iframe.cols, CV_8UC1);
	if (history.size() > hframe_count)
	{
		history.erase(history.begin());
		for (int i = 0; i < hframe_count; ++i)
		{
			rframe += history[i]/(hframe_count-i);
		}
	}
	return rframe;
}

int main(int argc,char** argv)
{
	const int padding = 50;
	const string dir = "C:\\Users\\timya\\Desktop\\203394129.mp4";
	BRIEF::rBRIEF extractor;
	Orb orb;
	Profiler profiler;
	VideoCapture cap; 	
	namedWindow("traj", WINDOW_NORMAL);
	resizeWindow("traj", 1280, 720);
	moveWindow("traj", 50, 50);
	cap.open(dir);
	if (!cap.isOpened())
	{
		return -1;
	}
	Mat frame;
	cap.read(frame);
	Mat grey = Mat(frame.rows, frame.cols, CV_8UC1);	
	int frameWidth = frame.cols, frameHeight = frame.rows;
	int frameSize = frameWidth*frameHeight;
	uchar** grey2d = convert2D(grey.data, frameWidth, frameHeight);
	cuArray<uchar> gpuInputBuffer(frameSize), gpuOutputBuffer(frameSize);
	BRIEF::BRIEF::Features features_old;
	
	for (int fc = 0; waitKey(1)==-1; ++fc)
	{
		if (!cap.read(frame))break;
		cvtColor(frame, grey, CV_BGR2GRAY);	
		gpuInputBuffer.upload(grey.data);
		
		profiler.Start();	
		std::thread first([&]
		{
			boxFilter(grey, grey, -1, Size(5, 5));
		});
		vector<float4> corners = orb.fast(gpuInputBuffer, gpuOutputBuffer, 30, 9, frameWidth, frameHeight);
		orb.computeOrientation(gpuInputBuffer, corners, frameWidth, frameHeight);
		first.join();
		profiler.Log("FAST");

		vector<Point2d> keypoints;
		vector<float> angles;
		for (int i = 0; i < corners.size(); ++i)
		{			
			keypoints.push_back(Point2d(corners[i].x, corners[i].y)); 
			angles.push_back(corners[i].z);
		}

		profiler.Start();
		BRIEF::BRIEF::Features features = extractor.extractFeature(grey2d, keypoints, angles, frameWidth, frameHeight);
		profiler.Log("BRIEF");

		Mat hf(frameHeight, frameWidth, CV_8UC1);
		profiler.Start();
		
		BRIEF::matcher::MultiLSHashTable ht;
		ht.InsertRange(features);	
		profiler.Log("Hash_Build");
		auto mpairs = ht.Hash_Match(features_old,30);

		profiler.Log("Hash_Match");
		for (auto v : mpairs)
			line(hf, v.first, v.second, Scalar(255), 0.5, cv::LineTypes::LINE_AA);
		features_old = features;
		profiler.Log("Render");
		
		cv::imshow("traj", grey+ renderTrajectory(hf));
		//profiler.Log("Display");
		profiler.Report();
	}
	return 0;
}