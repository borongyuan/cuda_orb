#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>
#include "Profiler.h"
#include "Memory.cuh"
#include "FAST.cuh"
#include <math.h>
#include "Orb.h"
using namespace cv;
using namespace std;


Mat renderTrajectory(Mat& iframe)
{
	const int hframe_count = 8;
	static vector<Mat> history;

	history.push_back(iframe);
	
	Mat rframe(iframe.rows, iframe.cols, CV_8UC1);
	if (history.size() > hframe_count)
	{

		history.erase(history.begin());
		for (int i = 0; i < hframe_count; ++i)
		{
			rframe += history[i];
		}
	}
	return rframe;
}

int main(int argc,char** argv)
{
	const int hframe_count = 9;
	const int padding = 50;
	rBRIEF extractor;
	Orb orb;
	Profiler profiler;
	VideoCapture cap; 	

	cap.open("\\\\140.118.7.213\\Dataset\\sequence\\3.mp4");
	if (!cap.isOpened())
	{
		return -1;
	}
	Mat frame;
	cap.read(frame);
	Mat grey = Mat(frame.rows, frame.cols, CV_8UC1);	
	int frameWidth = frame.cols, frameHeight = frame.rows;
	int frameSize = frameWidth*frameHeight;
	Mat i5 = Mat(frame.rows, frame.cols, CV_8UC1);
	uchar** grey2d = convert2D(i5.data, frameWidth, frameHeight);
	cuArray<uchar> gpuInputBuffer(frameSize);
	cuArray<uchar> gpuOutputBuffer(frameSize);
	cuArray<float4> AngleMap(CORNER_LIMIT);
	vector<Mat> history;
	BRIEF::Features features_old;

	for (int fc=0;;++fc)
	{
		if (!cap.read(frame))break;
		cvtColor(frame, grey, CV_BGR2GRAY);	
		gpuInputBuffer.upload(grey.data);
		

		profiler.Start();
		vector<float4> corners = orb.fast(gpuInputBuffer, gpuOutputBuffer, frameWidth, frameHeight);
		orb.computeOrientation(gpuInputBuffer, corners, frameWidth, frameHeight);
		profiler.Log("FAST+Orientation");

		profiler.Start();
		boxFilter(grey, i5, -1, Size(5, 5));
		profiler.Log("Blur");

		vector<Point2d> keypoints;
		vector<float> angles;
		for (int i = 0; i < corners.size(); ++i)
		{			
			keypoints.push_back(Point2d(corners[i].x, corners[i].y));
			angles.push_back(corners[i].z);
		}

		profiler.Log("Render");
		BRIEF::Features features = extractor.extractFeature(grey2d, keypoints, angles, grey.cols, grey.rows);
		profiler.Log("BRIEF");

		Mat hf(frameHeight, frameWidth, CV_8UC1);
		for (auto v: MatchBF(features, features_old))
			line(hf, v.first,v.second, Scalar(255, 255, 0), 1, cv::LineTypes::LINE_AA);

		profiler.Log("Match");
		features_old = features;
		profiler.Report(); 
	  	cv::imshow("output", frame);	
		cv::imshow("traj", renderTrajectory(hf));
		//waitKey();
		if (waitKey(1) >= 0) break;
	}
	return 0;
}