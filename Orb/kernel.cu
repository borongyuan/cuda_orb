#include "hip/hip_runtime.h"
﻿
#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>
#include <thread>
#include "Profiler.h"
#include "Memory.h"
#include "Orb.h"
using namespace cv;
using namespace std;


Mat renderTrajectory(Mat& iframe)
{
	const int hframe_count = 8;
	static vector<Mat> history;
	history.push_back(iframe);
	Mat rframe(iframe.rows, iframe.cols, CV_8UC1);
	if (history.size() > hframe_count)
	{
		history.erase(history.begin());
		for (int i = 0; i < hframe_count; ++i)
		{
			rframe += history[i]/(hframe_count-i);
		}
	}

	return rframe;
}

vector<Orb::Feature> TrackKeypoints(Mat& frame,Orb& orb)
{
	Mat grey;
	cvtColor(frame, grey, CV_BGR2GRAY);
	uchar** grey2d = convert2D(grey.data, grey.cols, grey.rows);
	vector<float4> corners = orb.detectKeypoints(grey, 25, 12, 1500);
	vector<Orb::Feature> features = orb.extractFeatures(grey2d, corners);
	return features;
}


int main(int argc,char** argv)
{
	Orb orb;
	BRIEF::Optimizer optimizer;
	Profiler profiler;
	VideoCapture cap; 	
	namedWindow("traj", WINDOW_NORMAL);
	resizeWindow("traj", 1280, 720);
	moveWindow("traj", 50, 50);
	cap.open(argv[1]);
	if (!cap.isOpened())
	{
		return -1;
	}
	Mat frame;
	Mat tframe = Mat(frame.rows,frame.cols,frame.type());
	vector<Orb::Feature> features_old;
	for (int fc = 0; waitKey(1)==-1; ++fc)
	{
		if (!cap.read(frame))break;
		vector<Orb::Feature> features = TrackKeypoints(frame, orb);
		for (vector<Orb::Feature>::iterator it = features.begin(); it < features.end(); ++it)
		{
			circle(frame, it->position, 2, Scalar(155, 255, 125), 1);
		}
		BRIEF::MultiLSHashTable hs;
		hs.InsertRange(features);
		if (features_old.size() > 0)
		{
			for (auto mp : hs.Hash_Match(features_old))
				line(frame, mp.first, mp.second, Scalar(255, 255, 225), 1, cv::LineTypes::LINE_AA);
		}
		features_old = features;
		imshow("traj",frame);
	}
	return 0;
}