#include "hip/hip_runtime.h"
﻿
#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>
#include <thread>
#include "Profiler.h"
#include "Memory.h"
#include "Orb.h"
#include <sstream>
#include <fstream>
using namespace cv;
using namespace std;


Mat renderTrajectory(Mat& iframe)
{
	const int hframe_count = 8;
	static vector<Mat> history;
	history.push_back(iframe);
	Mat rframe(iframe.rows, iframe.cols, iframe.type());
	if (history.size() > hframe_count)
	{
		history.erase(history.begin());
		for (int i = 0; i < hframe_count; ++i)
		{
			rframe += history[i]/(hframe_count-i);
		}
	}
	return rframe;
}

vector<Orb::Feature> TrackKeypoints(Mat& frame,Orb& orb)
{
	Mat grey;
	cvtColor(frame, grey, CV_BGR2GRAY);
	uchar** grey2d = convert2D(grey.data, grey.cols, grey.rows);
	vector<float4> corners = orb.detectKeypoints(grey, 25, 12, 1500);
	vector<Orb::Feature> features = orb.extractFeatures(grey2d, corners,Orb::MODE::MODE_RBRIEF);
	return features;
}

void TrackCamera(char* arg)
{
	Orb orb;
	BRIEF::Optimizer optimizer;
	Profiler profiler;
	VideoCapture cap;
	Mat frame;
	namedWindow("traj", WINDOW_NORMAL);
	resizeWindow("traj", 1280, 720);
	moveWindow("traj", 50, 50);
	cap.open(arg);
	if (!cap.isOpened())
		return;
	vector<Orb::Feature> features_old;
	for (int fc = 0; waitKey(1) == -1; ++fc)
	{
		if (!cap.read(frame))break;
		vector<Orb::Feature> features = TrackKeypoints(frame, orb);
		if (features_old.size() > 0)
		{
			BRIEF::MultiLSHashTable hs;
			hs.InsertRange(features);
			for (auto mp : hs.Hash_Match(features_old))
				line(frame, mp.first, mp.second, Scalar(255, 255, 225), 1, cv::LineTypes::LINE_AA);
		}
		features_old = features;
		imshow("traj", frame);
	}
}


void BRIEF_Optimize(char* p)
{	
	string path;
	fstream f(p);
	Orb orb;
	BRIEF::Optimizer optimizer;
	vector<Orb::Feature> features;
	for (;getline(f, path);)
	{
		cout << path << endl;
		Mat m = imread(path);
		imshow("n", m);
		waitKey(5);
		vector<Orb::Feature> fs = TrackKeypoints(m,orb);
		features.insert(features.end(), fs.begin(), fs.end());
	}
	f.close();
	cout << optimizer.computeVariance(features);
}

int main(int argc,char** argv)
{
	BRIEF_Optimize(argv[1]);
	
	return 0;
}